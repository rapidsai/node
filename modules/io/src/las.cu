#include "hip/hip_runtime.h"
// Copyright (c) 2021, NVIDIA CORPORATION.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <iostream>

#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>

#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <cudf/column/column_factories.hpp>
#include <cudf/io/datasource.hpp>

#include <las.hpp>

__global__ void parse_header(uint8_t const* las_header_data, LasHeader* result) {
  size_t byte_offset = 0;

  // File signature (4 bytes)
  for (int i = 0; i < 4; ++i) { result->file_signature[i] = *(las_header_data + i); }
  byte_offset += 4;

  // File source id (2 bytes)
  result->file_source_id = *(las_header_data + byte_offset) | *(las_header_data + byte_offset + 1)
                                                                << 8;
  byte_offset += 2;

  // Global encoding (2 bytes)
  result->global_encoding = *(las_header_data + byte_offset) | *(las_header_data + byte_offset + 1)
                                                                 << 8;
  byte_offset += 2;

  // Project ID (16 bytes)
  // not required
  byte_offset += 16;

  // Version major (1 byte)
  result->version_major = *(las_header_data + byte_offset);
  byte_offset += 1;

  // Version minor (1 byte)
  result->version_minor = *(las_header_data + byte_offset);
  byte_offset += 1;

  // System identifier (32 bytes)
  for (int i = 0; i < 32; ++i) {
    result->system_identifier[i] = *(las_header_data + byte_offset + i);
  }
  byte_offset += 32;

  // Generating software (32 bytes)
  for (int i = 0; i < 32; ++i) {
    result->generating_software[i] = *(las_header_data + byte_offset + i);
  }
  byte_offset += 32;

  // File creation day of year (2 bytes)
  // not required
  byte_offset += 2;

  // File creation year (2 bytes)
  // not required
  byte_offset += 2;

  // Header size (2 bytes)
  result->header_size = *(las_header_data + byte_offset) | *(las_header_data + byte_offset + 1)
                                                             << 8;
  byte_offset += 2;

  // Offset to point data (4 bytes)
  result->point_data_offset =
    *(las_header_data + byte_offset) | *(las_header_data + byte_offset + 1) << 8 |
    *(las_header_data + byte_offset + 2) << 16 | *(las_header_data + byte_offset + 3) << 24;
  byte_offset += 4;

  // Number of variable length records (4 bytes)
  result->variable_length_records_count =
    *(las_header_data + byte_offset) | *(las_header_data + byte_offset + 1) << 8 |
    *(las_header_data + byte_offset + 2) << 16 | *(las_header_data + byte_offset + 3) << 24;
  byte_offset += 4;

  // Point data format id (1 byte)
  result->point_data_format_id = *(las_header_data + byte_offset);
  if (result->point_data_format_id & 128 || result->point_data_format_id & 64)
    result->point_data_format_id &= 127;
  byte_offset += 1;

  // Point data record length (2 bytes)
  result->point_data_size = *(las_header_data + byte_offset) | *(las_header_data + byte_offset + 1)
                                                                 << 8;
  byte_offset += 2;

  // Number of point records (4 bytes)
  result->point_record_count =
    *(las_header_data + byte_offset) | *(las_header_data + byte_offset + 1) << 8 |
    *(las_header_data + byte_offset + 2) << 16 | *(las_header_data + byte_offset + 3) << 24;
  byte_offset += 4;

  // Number of points by return (20 bytes)
  for (int i = 0; i < 4; ++i) {
    result->points_by_return_count[i] =
      *(las_header_data + byte_offset) | *(las_header_data + byte_offset + 1) << 8 |
      *(las_header_data + byte_offset + 2) << 16 | *(las_header_data + byte_offset + 3) << 24;
    byte_offset += 4;
  }

  // X scale factor (8 bytes)
  result->x_scale =
    *(las_header_data + byte_offset) | *(las_header_data + byte_offset + 1) << 8 |
    *(las_header_data + byte_offset + 2) << 16 | *(las_header_data + byte_offset + 3) << 24 |
    *(las_header_data + byte_offset + 4) << 32 | *(las_header_data + byte_offset + 5) << 40 |
    *(las_header_data + byte_offset + 6) << 48 | *(las_header_data + byte_offset + 7) << 56;
  byte_offset += 8;

  // Y scale factor (8 bytes)
  result->y_scale =
    *(las_header_data + byte_offset) | *(las_header_data + byte_offset + 1) << 8 |
    *(las_header_data + byte_offset + 2) << 16 | *(las_header_data + byte_offset + 3) << 24 |
    *(las_header_data + byte_offset + 4) << 32 | *(las_header_data + byte_offset + 5) << 40 |
    *(las_header_data + byte_offset + 6) << 48 | *(las_header_data + byte_offset + 7) << 56;
  byte_offset += 8;

  // Z scale factor (8 bytes)
  result->z_scale =
    *(las_header_data + byte_offset) | *(las_header_data + byte_offset + 1) << 8 |
    *(las_header_data + byte_offset + 2) << 16 | *(las_header_data + byte_offset + 3) << 24 |
    *(las_header_data + byte_offset + 4) << 32 | *(las_header_data + byte_offset + 5) << 40 |
    *(las_header_data + byte_offset + 6) << 48 | *(las_header_data + byte_offset + 7) << 56;
  byte_offset += 8;

  // X offset (8 bytes)
  result->x_offset =
    *(las_header_data + byte_offset) | *(las_header_data + byte_offset + 1) << 8 |
    *(las_header_data + byte_offset + 2) << 16 | *(las_header_data + byte_offset + 3) << 24 |
    *(las_header_data + byte_offset + 4) << 32 | *(las_header_data + byte_offset + 5) << 40 |
    *(las_header_data + byte_offset + 6) << 48 | *(las_header_data + byte_offset + 7) << 56;
  byte_offset += 8;

  // Y offset (8 bytes)
  result->y_offset =
    *(las_header_data + byte_offset) | *(las_header_data + byte_offset + 1) << 8 |
    *(las_header_data + byte_offset + 2) << 16 | *(las_header_data + byte_offset + 3) << 24 |
    *(las_header_data + byte_offset + 4) << 32 | *(las_header_data + byte_offset + 5) << 40 |
    *(las_header_data + byte_offset + 6) << 48 | *(las_header_data + byte_offset + 7) << 56;
  byte_offset += 8;

  // Z offset (8 bytes)
  result->z_offset =
    *(las_header_data + byte_offset) | *(las_header_data + byte_offset + 1) << 8 |
    *(las_header_data + byte_offset + 2) << 16 | *(las_header_data + byte_offset + 3) << 24 |
    *(las_header_data + byte_offset + 4) << 32 | *(las_header_data + byte_offset + 5) << 40 |
    *(las_header_data + byte_offset + 6) << 48 | *(las_header_data + byte_offset + 7) << 56;
  byte_offset += 8;

  // Max X (8 bytes)
  result->max_x =
    *(las_header_data + byte_offset) | *(las_header_data + byte_offset + 1) << 8 |
    *(las_header_data + byte_offset + 2) << 16 | *(las_header_data + byte_offset + 3) << 24 |
    *(las_header_data + byte_offset + 4) << 32 | *(las_header_data + byte_offset + 5) << 40 |
    *(las_header_data + byte_offset + 6) << 48 | *(las_header_data + byte_offset + 7) << 56;
  byte_offset += 8;

  // Min X (8 bytes)
  result->min_x =
    *(las_header_data + byte_offset) | *(las_header_data + byte_offset + 1) << 8 |
    *(las_header_data + byte_offset + 2) << 16 | *(las_header_data + byte_offset + 3) << 24 |
    *(las_header_data + byte_offset + 4) << 32 | *(las_header_data + byte_offset + 5) << 40 |
    *(las_header_data + byte_offset + 6) << 48 | *(las_header_data + byte_offset + 7) << 56;
  byte_offset += 8;

  // Max Y (8 bytes)
  result->max_y =
    *(las_header_data + byte_offset) | *(las_header_data + byte_offset + 1) << 8 |
    *(las_header_data + byte_offset + 2) << 16 | *(las_header_data + byte_offset + 3) << 24 |
    *(las_header_data + byte_offset + 4) << 32 | *(las_header_data + byte_offset + 5) << 40 |
    *(las_header_data + byte_offset + 6) << 48 | *(las_header_data + byte_offset + 7) << 56;
  byte_offset += 8;

  // Min Y (8 bytes)
  result->min_y =
    *(las_header_data + byte_offset) | *(las_header_data + byte_offset + 1) << 8 |
    *(las_header_data + byte_offset + 2) << 16 | *(las_header_data + byte_offset + 3) << 24 |
    *(las_header_data + byte_offset + 4) << 32 | *(las_header_data + byte_offset + 5) << 40 |
    *(las_header_data + byte_offset + 6) << 48 | *(las_header_data + byte_offset + 7) << 56;
  byte_offset += 8;

  // Max Z (8 bytes)
  result->max_z =
    *(las_header_data + byte_offset) | *(las_header_data + byte_offset + 1) << 8 |
    *(las_header_data + byte_offset + 2) << 16 | *(las_header_data + byte_offset + 3) << 24 |
    *(las_header_data + byte_offset + 4) << 32 | *(las_header_data + byte_offset + 5) << 40 |
    *(las_header_data + byte_offset + 6) << 48 | *(las_header_data + byte_offset + 7) << 56;
  byte_offset += 8;

  // Min Z (8 bytes)
  result->min_z =
    *(las_header_data + byte_offset) | *(las_header_data + byte_offset + 1) << 8 |
    *(las_header_data + byte_offset + 2) << 16 | *(las_header_data + byte_offset + 3) << 24 |
    *(las_header_data + byte_offset + 4) << 32 | *(las_header_data + byte_offset + 5) << 40 |
    *(las_header_data + byte_offset + 6) << 48 | *(las_header_data + byte_offset + 7) << 56;
}

std::unique_ptr<cudf::io::datasource::buffer> read(
  const std::unique_ptr<cudf::io::datasource>& datasource,
  size_t offset,
  size_t size,
  rmm::cuda_stream_view stream) {
  if (datasource->supports_device_read()) { return datasource->device_read(offset, size, stream); }
  auto device_buffer = rmm::device_buffer(size, stream);
  CUDA_TRY(hipMemcpyAsync(device_buffer.data(),
                           datasource->host_read(offset, size)->data(),
                           size,
                           hipMemcpyHostToDevice,
                           stream.value()));
  return cudf::io::datasource::buffer::create(std::move(device_buffer));
}

std::unique_ptr<cudf::table> get_point_cloud_records(
  const std::unique_ptr<cudf::io::datasource>& datasource,
  LasHeader* cpu_header,
  rmm::mr::device_memory_resource* mr,
  rmm::cuda_stream_view stream) {
  auto const& point_record_count = cpu_header->point_record_count;
  auto const& point_data_offset  = cpu_header->point_data_offset;
  auto const& point_data_size    = cpu_header->point_data_size;

  auto point_data =
    read(datasource, point_data_offset, point_data_size * point_record_count, stream);

  auto data = point_data->data();
  auto idxs = thrust::make_counting_iterator(0);
  std::vector<std::unique_ptr<cudf::column>> cols;

  switch (cpu_header->point_data_format_id) {
    // POINT
    // FORMAT
    // ZERO
    case 0: {
      cols.resize(9);

      std::vector<cudf::type_id> ids{{
        cudf::type_id::INT32,  // x
        cudf::type_id::INT32,  // y
        cudf::type_id::INT32,  // z
        cudf::type_id::INT16,  // intensity
        cudf::type_id::INT8,   // bit_data
        cudf::type_id::INT8,   // classification
        cudf::type_id::INT8,   // scan angle
        cudf::type_id::INT8,   // user data
        cudf::type_id::INT16,  // point source id
      }};

      std::transform(ids.begin(), ids.end(), cols.begin(), [&](auto const& type_id) {
        return cudf::make_numeric_column(
          cudf::data_type{type_id}, point_record_count, cudf::mask_state::UNALLOCATED, stream, mr);
      });

      auto iter = thrust::make_transform_iterator(idxs, [=] __host__ __device__(int const& i) {
        auto ptr = data + (i * (point_data_size));
        PointDataFormatZero point_data;
        point_data.x               = static_cast<int32_t const>(*(ptr + 0));
        point_data.y               = static_cast<int32_t const>(*(ptr + 4));
        point_data.z               = static_cast<int32_t const>(*(ptr + 8));
        point_data.intensity       = static_cast<int16_t const>(*(ptr + 12));
        point_data.bit_data        = static_cast<int8_t const>(*(ptr + 14));
        point_data.classification  = static_cast<int8_t const>(*(ptr + 15));
        point_data.scan_angle      = static_cast<int8_t const>(*(ptr + 16));
        point_data.user_data       = static_cast<int8_t const>(*(ptr + 17));
        point_data.point_source_id = static_cast<int16_t const>(*(ptr + 18));
        return thrust::make_tuple(point_data.x,
                                  point_data.y,
                                  point_data.z,
                                  point_data.intensity,
                                  point_data.bit_data,
                                  point_data.classification,
                                  point_data.scan_angle,
                                  point_data.user_data,
                                  point_data.point_source_id);
      });

      thrust::copy(
        rmm::exec_policy(stream),
        iter,
        iter + point_record_count,
        thrust::make_zip_iterator(cols[0]->mutable_view().begin<int32_t>(),    // x
                                  cols[1]->mutable_view().begin<int32_t>(),    // y
                                  cols[2]->mutable_view().begin<int32_t>(),    // z
                                  cols[3]->mutable_view().begin<int16_t>(),    // intensity
                                  cols[4]->mutable_view().begin<int8_t>(),     // bits
                                  cols[5]->mutable_view().begin<int8_t>(),     // classification
                                  cols[6]->mutable_view().begin<int8_t>(),     // scan angle
                                  cols[7]->mutable_view().begin<int8_t>(),     // user data
                                  cols[8]->mutable_view().begin<int16_t>()));  // point source id
      break;
    }

    // POINT
    // FORMAT
    // ONE
    case 1: {
      cols.resize(10);

      std::vector<cudf::type_id> ids{{
        cudf::type_id::INT32,    // x
        cudf::type_id::INT32,    // y
        cudf::type_id::INT32,    // z
        cudf::type_id::INT16,    // intensity
        cudf::type_id::INT8,     // bit_data
        cudf::type_id::INT8,     // classification
        cudf::type_id::INT8,     // scan angle
        cudf::type_id::INT8,     // user data
        cudf::type_id::INT16,    // point source id
        cudf::type_id::FLOAT64,  // gps time
      }};

      std::transform(ids.begin(), ids.end(), cols.begin(), [&](auto const& type_id) {
        return cudf::make_numeric_column(
          cudf::data_type{type_id}, point_record_count, cudf::mask_state::UNALLOCATED, stream, mr);
      });

      auto iter = thrust::make_transform_iterator(idxs, [=] __host__ __device__(int const& i) {
        auto ptr = data + (i * (point_data_size));
        PointDataFormatOne point_data;
        point_data.x               = static_cast<int32_t const>(*(ptr + 0));
        point_data.y               = static_cast<int32_t const>(*(ptr + 4));
        point_data.z               = static_cast<int32_t const>(*(ptr + 8));
        point_data.intensity       = static_cast<int16_t const>(*(ptr + 12));
        point_data.bit_data        = static_cast<int8_t const>(*(ptr + 14));
        point_data.classification  = static_cast<int8_t const>(*(ptr + 15));
        point_data.scan_angle      = static_cast<int8_t const>(*(ptr + 16));
        point_data.user_data       = static_cast<int8_t const>(*(ptr + 17));
        point_data.point_source_id = static_cast<int16_t const>(*(ptr + 18));
        point_data.gps_time        = static_cast<double_t const>(*(ptr + 20));
        return thrust::make_tuple(point_data.x,
                                  point_data.y,
                                  point_data.z,
                                  point_data.intensity,
                                  point_data.bit_data,
                                  point_data.classification,
                                  point_data.scan_angle,
                                  point_data.user_data,
                                  point_data.point_source_id,
                                  point_data.gps_time);
      });

      thrust::copy(
        rmm::exec_policy(stream),
        iter,
        iter + point_record_count,
        thrust::make_zip_iterator(cols[0]->mutable_view().begin<int32_t>(),     // x
                                  cols[1]->mutable_view().begin<int32_t>(),     // y
                                  cols[2]->mutable_view().begin<int32_t>(),     // z
                                  cols[3]->mutable_view().begin<int16_t>(),     // intensity
                                  cols[4]->mutable_view().begin<int8_t>(),      // bits
                                  cols[5]->mutable_view().begin<int8_t>(),      // classification
                                  cols[6]->mutable_view().begin<int8_t>(),      // scan angle
                                  cols[7]->mutable_view().begin<int8_t>(),      // user data
                                  cols[8]->mutable_view().begin<int16_t>(),     // point source id
                                  cols[9]->mutable_view().begin<double_t>()));  // gps time
      break;
    }

    // POINT
    // FORMAT
    // THREE
    // TODO: Missing colours
    case 2: {
      cols.resize(9);

      std::vector<cudf::type_id> ids{{
        cudf::type_id::INT32,  // x
        cudf::type_id::INT32,  // y
        cudf::type_id::INT32,  // z
        cudf::type_id::INT16,  // intensity
        cudf::type_id::INT8,   // bit_data
        cudf::type_id::INT8,   // classification
        cudf::type_id::INT8,   // scan angle
        cudf::type_id::INT8,   // user data
        cudf::type_id::INT16,  // point source id
      }};

      std::transform(ids.begin(), ids.end(), cols.begin(), [&](auto const& type_id) {
        return cudf::make_numeric_column(
          cudf::data_type{type_id}, point_record_count, cudf::mask_state::UNALLOCATED, stream, mr);
      });

      auto iter = thrust::make_transform_iterator(idxs, [=] __host__ __device__(int const& i) {
        auto ptr = data + (i * (point_data_size));
        PointDataFormatTwo point_data;
        point_data.x               = static_cast<int32_t const>(*(ptr + 0));
        point_data.y               = static_cast<int32_t const>(*(ptr + 4));
        point_data.z               = static_cast<int32_t const>(*(ptr + 8));
        point_data.intensity       = static_cast<int16_t const>(*(ptr + 12));
        point_data.bit_data        = static_cast<int8_t const>(*(ptr + 14));
        point_data.classification  = static_cast<int8_t const>(*(ptr + 15));
        point_data.scan_angle      = static_cast<int8_t const>(*(ptr + 16));
        point_data.user_data       = static_cast<int8_t const>(*(ptr + 17));
        point_data.point_source_id = static_cast<int16_t const>(*(ptr + 18));
        return thrust::make_tuple(point_data.x,
                                  point_data.y,
                                  point_data.z,
                                  point_data.intensity,
                                  point_data.bit_data,
                                  point_data.classification,
                                  point_data.scan_angle,
                                  point_data.user_data,
                                  point_data.point_source_id);
      });

      thrust::copy(
        rmm::exec_policy(stream),
        iter,
        iter + point_record_count,
        thrust::make_zip_iterator(cols[0]->mutable_view().begin<int32_t>(),    // x
                                  cols[1]->mutable_view().begin<int32_t>(),    // y
                                  cols[2]->mutable_view().begin<int32_t>(),    // z
                                  cols[3]->mutable_view().begin<int16_t>(),    // intensity
                                  cols[4]->mutable_view().begin<int8_t>(),     // bits
                                  cols[5]->mutable_view().begin<int8_t>(),     // classification
                                  cols[6]->mutable_view().begin<int8_t>(),     // scan angle
                                  cols[7]->mutable_view().begin<int8_t>(),     // user data
                                  cols[8]->mutable_view().begin<int16_t>()));  // point source id
      break;
    }

    // POINT
    // FORMAT
    // THREE
    // TODO: Missing colours
    case 3: {
      cols.resize(10);

      std::vector<cudf::type_id> ids{{
        cudf::type_id::INT32,    // x
        cudf::type_id::INT32,    // y
        cudf::type_id::INT32,    // z
        cudf::type_id::INT16,    // intensity
        cudf::type_id::INT8,     // bit_data
        cudf::type_id::INT8,     // classification
        cudf::type_id::INT8,     // scan angle
        cudf::type_id::INT8,     // user data
        cudf::type_id::INT16,    // point source id
        cudf::type_id::FLOAT64,  // gps time
      }};

      std::transform(ids.begin(), ids.end(), cols.begin(), [&](auto const& type_id) {
        return cudf::make_numeric_column(
          cudf::data_type{type_id}, point_record_count, cudf::mask_state::UNALLOCATED, stream, mr);
      });

      auto iter = thrust::make_transform_iterator(idxs, [=] __host__ __device__(int const& i) {
        auto ptr = data + (i * (point_data_size));
        PointDataFormatThree point_data;
        point_data.x               = static_cast<int32_t const>(*(ptr + 0));
        point_data.y               = static_cast<int32_t const>(*(ptr + 4));
        point_data.z               = static_cast<int32_t const>(*(ptr + 8));
        point_data.intensity       = static_cast<int16_t const>(*(ptr + 12));
        point_data.bit_data        = static_cast<int8_t const>(*(ptr + 14));
        point_data.classification  = static_cast<int8_t const>(*(ptr + 15));
        point_data.scan_angle      = static_cast<int8_t const>(*(ptr + 16));
        point_data.user_data       = static_cast<int8_t const>(*(ptr + 17));
        point_data.point_source_id = static_cast<int16_t const>(*(ptr + 18));
        point_data.gps_time        = static_cast<double_t const>(*(ptr + 20));
        return thrust::make_tuple(point_data.x,
                                  point_data.y,
                                  point_data.z,
                                  point_data.intensity,
                                  point_data.bit_data,
                                  point_data.classification,
                                  point_data.scan_angle,
                                  point_data.user_data,
                                  point_data.point_source_id,
                                  point_data.gps_time);
      });

      thrust::copy(
        rmm::exec_policy(stream),
        iter,
        iter + point_record_count,
        thrust::make_zip_iterator(cols[0]->mutable_view().begin<int32_t>(),     // x
                                  cols[1]->mutable_view().begin<int32_t>(),     // y
                                  cols[2]->mutable_view().begin<int32_t>(),     // z
                                  cols[3]->mutable_view().begin<int16_t>(),     // intensity
                                  cols[4]->mutable_view().begin<int8_t>(),      // bits
                                  cols[5]->mutable_view().begin<int8_t>(),      // classification
                                  cols[6]->mutable_view().begin<int8_t>(),      // scan angle
                                  cols[7]->mutable_view().begin<int8_t>(),      // user data
                                  cols[8]->mutable_view().begin<int16_t>(),     // point source id
                                  cols[9]->mutable_view().begin<double_t>()));  // gps time
      break;
    }
  }

  return std::make_unique<cudf::table>(std::move(cols));
}

void parse_las_header_host(const std::unique_ptr<cudf::io::datasource>& datasource,
                           LasHeader* cpu_header,
                           LasHeader* gpu_header,
                           rmm::cuda_stream_view stream) {
  auto header_data = read(datasource, 0, HEADER_BYTE_SIZE, stream);
  ::parse_header<<<1, 1>>>(header_data->data(), gpu_header);

  hipMemcpy(cpu_header, gpu_header, sizeof(LasHeader), hipMemcpyDeviceToHost);
}

std::unique_ptr<cudf::table> parse_las_host(const std::unique_ptr<cudf::io::datasource>& datasource,
                                            rmm::mr::device_memory_resource* mr,
                                            rmm::cuda_stream_view stream) {
  LasHeader *cpu_header, *gpu_header;
  cpu_header = (LasHeader*)malloc(sizeof(LasHeader));
  hipMalloc((void**)&gpu_header, sizeof(LasHeader));
  parse_las_header_host(datasource, cpu_header, gpu_header, stream);

  auto table = get_point_cloud_records(datasource, cpu_header, mr, stream);

  free(cpu_header);
  hipFree(gpu_header);

  return table;
}
