#include "hip/hip_runtime.h"
// Copyright (c) 2021, NVIDIA CORPORATION.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <iostream>
#include <laz.hpp>

#include <cudf/io/datasource.hpp>

__global__ void parse_header(uint8_t const* laz_header_data, LazHeader* result) {
  size_t byte_offset = 0;

  // File signature (4 bytes)
  for (int i = byte_offset; i < 4; ++i) { result->file_signature[i] = *(laz_header_data + i); }
  byte_offset += 4;

  // File source id (2 bytes)
  result->file_source_id = *(laz_header_data + byte_offset) + *(laz_header_data + byte_offset + 1);
  byte_offset += 2;

  // Global encoding (2 bytes)
  result->global_encoding = *(laz_header_data + byte_offset) + *(laz_header_data + byte_offset + 1);
  byte_offset += 2;

  // Project ID (16 bytes)
  // not required
  byte_offset += 16;

  // Version major (1 byte)
  result->version_major = *(laz_header_data + byte_offset);
  byte_offset += 1;

  // Version minor (1 byte)
  result->version_minor = *(laz_header_data + byte_offset);
  byte_offset += 1;

  // System identifier (32 bytes)
  for (int i = 0; i < 32; ++i) {
    result->system_identifier[i] = *(laz_header_data + byte_offset + i);
  }
  byte_offset += 32;

  // Generating software (32 bytes)
  for (int i = 0; i < 32; ++i) {
    result->generating_software[i] = *(laz_header_data + byte_offset + i);
  }
  byte_offset += 32;

  // File creation day of year (2 bytes)
  // not required
  byte_offset += 2;

  // File creation year (2 bytes)
  // not required
  byte_offset += 2;

  // Header size (2 byes)
  result->header_size = *(laz_header_data + byte_offset) + *(laz_header_data + byte_offset + 1);
  byte_offset += 2;

  // Offset to point data (4 bytes)
  result->point_data_offset =
    *(laz_header_data + byte_offset) + *(laz_header_data + byte_offset + 1) +
    *(laz_header_data + byte_offset + 2) + *(laz_header_data + byte_offset + 3);
  byte_offset += 4;

  // Number of variable length records (4 bytes)
  result->variable_length_records_count =
    *(laz_header_data + byte_offset) + *(laz_header_data + byte_offset + 1) +
    *(laz_header_data + byte_offset + 2) + *(laz_header_data + byte_offset + 3);
  byte_offset += 4;

  // Point data format id (1 byte)
  result->point_data_format_id = *(laz_header_data + byte_offset);
  byte_offset += 1;

  // Point data record length (2 bytes)
  result->point_data_record_length =
    *(laz_header_data + byte_offset) + *(laz_header_data + byte_offset + 1);
  byte_offset += 2;

  // Number of point records (4 bytes)
  result->point_record_count =
    *(laz_header_data + byte_offset) + *(laz_header_data + byte_offset + 1) +
    *(laz_header_data + byte_offset + 2) + *(laz_header_data + byte_offset + 3);
  byte_offset += 4;

  // Number of points by return (20 bytes)
  for (int i = 0; i < 4; ++i) {
    result->points_by_return_count[i] =
      *(laz_header_data + byte_offset) + *(laz_header_data + byte_offset + 1) +
      *(laz_header_data + byte_offset + 2) + *(laz_header_data + byte_offset + 3);
    byte_offset += 4;
  }

  // X scale factor (8 bytes)
  result->x_scale = *(laz_header_data + byte_offset) + *(laz_header_data + byte_offset + 1) +
                    *(laz_header_data + byte_offset + 2) + *(laz_header_data + byte_offset + 3) +
                    *(laz_header_data + byte_offset + 4) + *(laz_header_data + byte_offset + 5) +
                    *(laz_header_data + byte_offset + 6) + *(laz_header_data + byte_offset + 7);
  byte_offset += 8;

  // Y scale factor (8 bytes)
  result->y_scale = *(laz_header_data + byte_offset) + *(laz_header_data + byte_offset + 1) +
                    *(laz_header_data + byte_offset + 2) + *(laz_header_data + byte_offset + 3) +
                    *(laz_header_data + byte_offset + 4) + *(laz_header_data + byte_offset + 5) +
                    *(laz_header_data + byte_offset + 6) + *(laz_header_data + byte_offset + 7);
  byte_offset += 8;

  // Z scale factor (8 bytes)
  result->z_scale = *(laz_header_data + byte_offset) + *(laz_header_data + byte_offset + 1) +
                    *(laz_header_data + byte_offset + 2) + *(laz_header_data + byte_offset + 3) +
                    *(laz_header_data + byte_offset + 4) + *(laz_header_data + byte_offset + 5) +
                    *(laz_header_data + byte_offset + 6) + *(laz_header_data + byte_offset + 7);
  byte_offset += 8;

  // X offset (8 bytes)
  result->x_offset = *(laz_header_data + byte_offset) + *(laz_header_data + byte_offset + 1) +
                     *(laz_header_data + byte_offset + 2) + *(laz_header_data + byte_offset + 3) +
                     *(laz_header_data + byte_offset + 4) + *(laz_header_data + byte_offset + 5) +
                     *(laz_header_data + byte_offset + 6) + *(laz_header_data + byte_offset + 7);
  byte_offset += 8;

  // Y offset (8 bytes)
  result->y_offset = *(laz_header_data + byte_offset) + *(laz_header_data + byte_offset + 1) +
                     *(laz_header_data + byte_offset + 2) + *(laz_header_data + byte_offset + 3) +
                     *(laz_header_data + byte_offset + 4) + *(laz_header_data + byte_offset + 5) +
                     *(laz_header_data + byte_offset + 6) + *(laz_header_data + byte_offset + 7);
  byte_offset += 8;

  // Z offset (8 bytes)
  result->z_offset = *(laz_header_data + byte_offset) + *(laz_header_data + byte_offset + 1) +
                     *(laz_header_data + byte_offset + 2) + *(laz_header_data + byte_offset + 3) +
                     *(laz_header_data + byte_offset + 4) + *(laz_header_data + byte_offset + 5) +
                     *(laz_header_data + byte_offset + 6) + *(laz_header_data + byte_offset + 7);
  byte_offset += 8;

  // Max X (8 bytes)
  result->max_x = *(laz_header_data + byte_offset) + *(laz_header_data + byte_offset + 1) +
                  *(laz_header_data + byte_offset + 2) + *(laz_header_data + byte_offset + 3) +
                  *(laz_header_data + byte_offset + 4) + *(laz_header_data + byte_offset + 5) +
                  *(laz_header_data + byte_offset + 6) + *(laz_header_data + byte_offset + 7);
  byte_offset += 8;

  // Min X (8 bytes)
  result->min_x = *(laz_header_data + byte_offset) + *(laz_header_data + byte_offset + 1) +
                  *(laz_header_data + byte_offset + 2) + *(laz_header_data + byte_offset + 3) +
                  *(laz_header_data + byte_offset + 4) + *(laz_header_data + byte_offset + 5) +
                  *(laz_header_data + byte_offset + 6) + *(laz_header_data + byte_offset + 7);
  byte_offset += 8;

  // Max Y (8 bytes)
  result->max_y = *(laz_header_data + byte_offset) + *(laz_header_data + byte_offset + 1) +
                  *(laz_header_data + byte_offset + 2) + *(laz_header_data + byte_offset + 3) +
                  *(laz_header_data + byte_offset + 4) + *(laz_header_data + byte_offset + 5) +
                  *(laz_header_data + byte_offset + 6) + *(laz_header_data + byte_offset + 7);
  byte_offset += 8;

  // Min Y (8 bytes)
  result->min_y = *(laz_header_data + byte_offset) + *(laz_header_data + byte_offset + 1) +
                  *(laz_header_data + byte_offset + 2) + *(laz_header_data + byte_offset + 3) +
                  *(laz_header_data + byte_offset + 4) + *(laz_header_data + byte_offset + 5) +
                  *(laz_header_data + byte_offset + 6) + *(laz_header_data + byte_offset + 7);
  byte_offset += 8;

  // Max Z (8 bytes)
  result->max_z = *(laz_header_data + byte_offset) + *(laz_header_data + byte_offset + 1) +
                  *(laz_header_data + byte_offset + 2) + *(laz_header_data + byte_offset + 3) +
                  *(laz_header_data + byte_offset + 4) + *(laz_header_data + byte_offset + 5) +
                  *(laz_header_data + byte_offset + 6) + *(laz_header_data + byte_offset + 7);
  byte_offset += 8;

  // Min Z (8 bytes)
  result->min_z = *(laz_header_data + byte_offset) + *(laz_header_data + byte_offset + 1) +
                  *(laz_header_data + byte_offset + 2) + *(laz_header_data + byte_offset + 3) +
                  *(laz_header_data + byte_offset + 4) + *(laz_header_data + byte_offset + 5) +
                  *(laz_header_data + byte_offset + 6) + *(laz_header_data + byte_offset + 7);
}

void Laz::parse_header_host() {
  const size_t header_size = 227;
  auto header_data         = read(0, header_size, rmm::cuda_stream_default);

  LazHeader *cpu_header, *gpu_header;
  cpu_header = (LazHeader*)malloc(sizeof(LazHeader));
  hipMalloc((void**)&gpu_header, sizeof(LazHeader));

  ::parse_header<<<1, 1>>>(header_data->data(), gpu_header);

  hipMemcpy(cpu_header, gpu_header, sizeof(LazHeader), hipMemcpyDeviceToHost);

  free(cpu_header);
  hipFree(gpu_header);

  throw std::invalid_argument("end test");
}

std::unique_ptr<cudf::io::datasource::buffer> Laz::read(size_t offset,
                                                        size_t size,
                                                        rmm::cuda_stream_view stream) {
  if (_datasource->supports_device_read()) {
    return _datasource->device_read(offset, size, stream);
  }
  auto device_buffer = rmm::device_buffer(size, stream);
  CUDA_TRY(hipMemcpyAsync(device_buffer.data(),
                           _datasource->host_read(offset, size)->data(),
                           size,
                           hipMemcpyHostToDevice,
                           stream.value()));
  return cudf::io::datasource::buffer::create(std::move(device_buffer));
}
